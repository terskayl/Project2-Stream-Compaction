#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        void printArray(int n, int* a, bool abridged = false) {
            printf("    [ ");
            for (int i = 0; i < n; i++) {
                if (abridged && i + 2 == 15 && n > 16) {
                    i = n - 2;
                    printf("... ");
                }
                printf("%3d ", a[i]);
            }
            printf("]\n");
        }
        __global__ void kernUpsweepStep(int n, int exp, int* data) {
            unsigned idx = blockIdx.x * blockDim.x + threadIdx.x;
            if (idx >= n) return;

            // Similar to array modifications when making a heap, we want 
            // our array to be 1-indexed instead of 0-indexed because 
            // 0 is divisible by all powers of two - but we want that position
            // on the right.
            idx += 1;
            unsigned lowerNeighbor = idx - (1 << (exp - 1));
            if (idx <= n && lowerNeighbor >= 1 && idx % (1 << exp) == 0) {
                //data[idx - 1] += data[idx - (1 << (exp - 1)) - 1];
                data[idx - 1] += data[lowerNeighbor - 1];
            }
        }

        __global__ void kernDownsweepStep(int n, int exp, int* data) {
            unsigned idx = blockIdx.x * blockDim.x + threadIdx.x;
            if (idx >= n) return;

            // Similar to array modifications when making a heap, we want 
            // our array to be 1-indexed instead of 0-indexed because 
            // 0 is divisible by all powers of two - but we want that position
            // on the right.
            idx += 1;
            unsigned lowerNeighbor = idx - (1 << (exp - 1));
            if (idx <= n && lowerNeighbor >= 1 && idx % (1 << exp) == 0) {
                int temp = data[idx - 1];
                data[idx - 1] += data[lowerNeighbor - 1];
                data[lowerNeighbor - 1] = temp;
            }
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            timer().startGpuTimer();
            unsigned blocksize = 128;

            // n rounded up to the nearest power of two
            int roundUpN = ilog2ceil(n);
            int totalN = pow(2, roundUpN);
            
            int* d_data;
            hipMalloc((void**)&d_data, totalN * sizeof(int));
            checkCUDAError("hipMalloc d_data"); 

            hipMemset(d_data, 0, totalN * sizeof(int));
            checkCUDAError("hipMemset d_data");
            hipMemcpy(d_data, idata, n * sizeof(int), hipMemcpyHostToDevice);
            checkCUDAError("hipMemcpy initial data to d_data");

            // Up-Sweep
            for (int exp = 1; exp <= totalN; ++exp) {
                kernUpsweepStep<<<divup(totalN, blocksize), blocksize>>>(totalN, exp, d_data);
            }
            hipDeviceSynchronize();
            // Down-Sweep
            hipMemset(d_data + (totalN - 1), 0, 1 * sizeof(int));
            for (int exp = totalN; exp >= 1; --exp) {
                kernDownsweepStep<<<divup(totalN, blocksize), blocksize>>>(totalN, exp, d_data);
            }

            hipMemcpy(odata, d_data, n * sizeof(int), hipMemcpyDeviceToHost);
            checkCUDAError("hipMemcpy output data from d_data");
            hipFree(d_data);

            timer().endGpuTimer();
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
            timer().startGpuTimer();
            unsigned blocksize = 128;

            // n rounded up to the nearest power of two
            int roundUpN = ilog2ceil(n);
            int totalN = pow(2, roundUpN);

            int *d_data, *d_bools, *d_indices, *d_output;
            hipMalloc((void**)&d_data, totalN * sizeof(int));
            checkCUDAError("hipMalloc d_data");
            hipMalloc((void**)&d_bools, totalN * sizeof(int));
            checkCUDAError("hipMalloc d_bools");
            hipMalloc((void**)&d_indices, totalN * sizeof(int));
            checkCUDAError("hipMalloc d_indices");
            hipMalloc((void**)&d_output, totalN * sizeof(int));
            checkCUDAError("hipMalloc d_output");

            hipMemset(d_data, 0, totalN * sizeof(int));
            checkCUDAError("hipMemset d_data");
            hipMemcpy(d_data, idata, n * sizeof(int), hipMemcpyHostToDevice);
            checkCUDAError("hipMemcpy idata into d_data");

            Common::kernMapToBoolean<<<divup(totalN, blocksize), blocksize>>>(totalN, d_bools, d_data);
            checkCUDAError("kernMapToBoolean");
            hipDeviceSynchronize();
            // Up-Sweep
            for (int exp = 1; exp <= totalN; ++exp) {
                kernUpsweepStep << <divup(totalN, blocksize), blocksize >> > (totalN, exp, d_bools);
            }
            hipDeviceSynchronize();
            // Down-Sweep
            hipMemset(d_bools + (totalN - 1), 0, 1 * sizeof(int));
            for (int exp = totalN; exp >= 1; --exp) {
                kernDownsweepStep<<<divup(totalN, blocksize), blocksize>>>(totalN, exp, d_bools);
            }
            checkCUDAError("upsweep and downsweep scan");
            int sizePlusMaybeOne;
            hipMemcpy(&sizePlusMaybeOne, d_bools + (totalN - 1), 1 * sizeof(int), hipMemcpyDeviceToHost);
            checkCUDAError("hipMemcpy sizePlusMaybeOne");
            printf("%i", sizePlusMaybeOne);

            hipMemset(d_output, 0, totalN * sizeof(int));
            checkCUDAError("hipMemset d_output");

            Common::kernScatter<<<divup(totalN, blocksize), blocksize>>>(totalN, d_output, d_data, d_data, d_bools);
            checkCUDAError("kernScatter");
            hipMemcpy(odata, d_output, std::min(sizePlusMaybeOne, n) * sizeof(int), hipMemcpyDeviceToHost);
            checkCUDAError("hipMemcpy to output");

            if (odata[sizePlusMaybeOne - 1] == 0) {
                sizePlusMaybeOne--;
            }

            hipFree(d_data);
            hipFree(d_bools);
            hipFree(d_indices);
            hipFree(d_output);
            checkCUDAError("hipFree");


            timer().endGpuTimer();
            return sizePlusMaybeOne;
        }
    }
}
