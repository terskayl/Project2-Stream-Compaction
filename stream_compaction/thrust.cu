#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include "common.h"
#include "thrust.h"

namespace StreamCompaction {
    namespace Thrust {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            
            int *d_idata, *d_odata;
            hipMalloc((void**)&d_idata, n * sizeof(int));
            checkCUDAError("hipMalloc d_idata");
            hipMalloc((void**)&d_odata, n * sizeof(int));
            checkCUDAError("hipMalloc d_odata");

            hipMemcpy(d_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);
            checkCUDAError("hipMemcpy idata into d_idata");
            
            thrust::device_ptr<int> d_thrust_idata = thrust::device_ptr<int>(d_idata);
            thrust::device_ptr<int> d_thrust_odata = thrust::device_ptr<int>(d_odata);
            timer().startGpuTimer();
            // TODO use `thrust::exclusive_scan`
            // example: for device_vectors dv_in and dv_out:
            // thrust::exclusive_scan(dv_in.begin(), dv_in.end(), dv_out.begin());
            
            thrust::exclusive_scan(d_thrust_idata, d_thrust_idata + n, d_thrust_odata);

            timer().endGpuTimer();

            hipMemcpy(odata, d_odata, n * sizeof(int), hipMemcpyDeviceToHost);
            checkCUDAError("hipMemcpy d_odata into odata");
        }
    }
}
